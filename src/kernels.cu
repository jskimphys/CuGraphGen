#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hiprand.h>
#include <vector>
#include <thread>

#include "kernels.cuh"
#include "constants.h"

const int blockSize = 256;
const int STREAM_PER_WORKER = 16;

__global__ void generate_randombits_dst(vid_t prefix, uint16_t bab, uint16_t dcd, int num_bits, uint16_t* random_array, vid_t* output_array, uint64_t num_edges) {
    //inputs : after the bits in prefix, posterier bits are randomly generated
    //one_prob : probability of 1
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= num_edges) return;

    uint64_t rarr_idx = tid * num_bits;
    while(num_bits-- > 0) {
        if(output_array[2*tid] & (1 << num_bits)){//if the corresponding src_vid bit is 1
            if(random_array[rarr_idx++] < dcd){
                prefix = prefix | (1 << num_bits);
            }
        }
        else{
            if(random_array[rarr_idx++] < bab){
                prefix = prefix | (1 << num_bits);
            }
        }
    }
    output_array[2*tid+1] = prefix;
}

__global__ void generate_randombits_src(uint64_t prefix, uint16_t cdabcd, int num_bits, uint16_t* random_array, vid_t* output_array, uint64_t num_edges) {
    //inputs : after the bits in prefix, posterier bits are randomly generated
    //cdabcd : probability of 1
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= num_edges) return;

    uint64_t rarr_idx = tid * num_bits;
    while(num_bits-- >= 0) {
        if(random_array[rarr_idx++] < cdabcd){
            prefix = prefix | (1 << num_bits);
        }
    }
    output_array[2*tid] = prefix;
}

__global__ void fillWithStride2(vid_t* data, vid_t value, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        data[tid * 2] = value;
    }
}

CuWorker::CuWorker(uint64_t seed)
    : hostMemIdx(0), writer()
{
    size_t free_mem_size, total_mem_size;
    hipMemGetInfo(&free_mem_size, &total_mem_size);
    
    //allocate memory for each worker
    size_t mem_per_worker = (size_t) (free_mem_size * 0.85);
    double rarr_earr_ratio = 6;

    const int host_mem_num = 2;//this is for writing parallelism
    earr_bytesize = mem_per_worker / (rarr_earr_ratio + 1);
    rarr_bytesize = mem_per_worker - earr_bytesize;

    for(int i = 0; i < STREAM_PER_WORKER; i++){
        hipStream_t stream;
        hipStreamCreate(&stream);
        streams.push_back(stream);
    }

    hipMallocAsync((void**)&random_arr, rarr_bytesize + 8*1024, streams[0]);//extra 8KB for alignment(since wrongly aligned memory cause error)
    hipMallocAsync((void**)&edge_arr_device,  earr_bytesize + 8*1024, streams[1]);
    
    
    edge_arr_host_list = std::vector<vid_t*>(host_mem_num);
    for(int i=0; i< host_mem_num; i++){
        hipHostMalloc((void**)&(edge_arr_host_list[i]), earr_bytesize);
        if(hipPeekAtLastError() != hipSuccess){
            std::cerr << "Error in allocating memories" << std::endl;
            std::cerr << hipGetErrorString(hipPeekAtLastError()) << "at " << i << "th host memory allocation" << std::endl;
            exit(1);
        }
    }

    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);


    if(hipPeekAtLastError() != hipSuccess){
        std::cerr << "Error in allocating memories" << std::endl;
        std::cerr << hipGetErrorString(hipPeekAtLastError()) << std::endl;
        exit(1);
    }
}

CuWorker::~CuWorker(){
    for(int i=0; i< edge_arr_host_list.size(); i++){
        hipHostFree(edge_arr_host_list[i]);
    }
    hipFree(random_arr);
    hipFree(edge_arr_device);
    hiprandDestroyGenerator(gen);
    if(hipPeekAtLastError() != hipSuccess){
        std::cerr << "Error in freeing memory or destroying rand generator" << std::endl;
        std::cerr << hipGetErrorString(hipPeekAtLastError()) << std::endl;
        exit(1);
    }
    for(int i = 0; i < STREAM_PER_WORKER; i++){
        hipStreamDestroy(streams[i]);
    }
}

void CuWorker::update_random_arr(size_t num_32bits){
    hiprandGenerate(gen, random_arr, num_32bits);
    if(hipPeekAtLastError() != hipSuccess){
        std::cerr << "Error in generating random bits" << std::endl;
        std::cerr << hipGetErrorString(hipPeekAtLastError()) << std::endl;
        exit(1);
    }
}

void CuWorker::process_workloads(std::vector<schedule_entry> workloads, std::string filename, size_t filesize, double a, double b, double c, double d){

    size_t total_rbits = 0;
    size_t total_edges = 0;
    //caculate the total workload size
    for(auto entry : workloads){
        if(entry.t == schedule_entry::type::along_src_vid){
            total_rbits += entry.num_edge * (2*entry.log_n - entry.log_prefixlen);
        }
        else{
            total_rbits += entry.num_edge * (entry.log_n - entry.log_prefixlen);
        }
        total_edges += entry.num_edge;
    }
    update_random_arr(rarr_bytesize/4);//require 2 bytes of random value to generate one random bit in edge

    // maps memory for each workload
    vid_t** edge_ptrs = new vid_t*[workloads.size()];
    uint16_t** randombits_ptrs = new uint16_t*[workloads.size()];
    edge_ptrs[0] = (vid_t*)edge_arr_device;
    randombits_ptrs[0] = (uint16_t*)random_arr;
    for(int i = 1; i < workloads.size(); i++){
        edge_ptrs[i] = edge_ptrs[i-1] + workloads[i-1].num_edge * 2;
        edge_ptrs[i] = edge_ptrs[i] + 16 - (workloads[i-1].num_edge*2) % 16;//align to 16 byte

        size_t randombits_needed = 0;
        if(workloads[i].t == schedule_entry::type::along_src_vid){
            //need to generate random bits for src_vid
            randombits_needed = workloads[i-1].num_edge * (workloads[i-1].log_n - workloads[i-1].log_prefixlen);
            randombits_ptrs[i] = randombits_ptrs[i-1] + randombits_needed;
            randombits_ptrs[i] = randombits_ptrs[i] + 16 - randombits_needed % 16;//align to 16 byte
            
            //need to generate random bits for dst_vid
            randombits_needed = workloads[i-1].num_edge * workloads[i-1].log_n;
            randombits_ptrs[i] = randombits_ptrs[i] + randombits_needed;
            randombits_ptrs[i] = randombits_ptrs[i] + 16 - randombits_needed % 16;//align to 16 byte
        }
        else{
            randombits_needed = workloads[i-1].num_edge * (workloads[i-1].log_n - workloads[i-1].log_prefixlen);
            randombits_ptrs[i] = randombits_ptrs[i-1] + randombits_needed;
            randombits_ptrs[i] = randombits_ptrs[i] + 16 - randombits_needed % 16;//align to 16 byte
        }
    }

    #pragma omp parallel for
    for(int i = 0; i < workloads.size(); i++){

        schedule_entry& entry = workloads[i];
        if(entry.t == schedule_entry::type::along_dst_vid){
            //fill the src_vid in the edgelist
            uint64_t gridSize = (entry.num_edge + blockSize - 1) / blockSize;
            uint16_t bab = (uint16_t) round(b/(a+b) * (1 << 16) - 0.5);//-0.5 is here, becuase prob is distibuted from 0 to 2^16 but random uint16 is distributed from 0 to 2^16-1
            uint16_t dcd = (uint16_t) round(d/(c+d) * (1 << 16) - 0.5);

            fillWithStride2<<<gridSize, blockSize, 0, streams[i % streams.size()]>>>(edge_ptrs[i], entry.src_vid_start, entry.num_edge);
            generate_randombits_dst<<<gridSize, blockSize, 0, streams[i % streams.size()]>>>(entry.dst_vid_start, bab, dcd, entry.log_n - entry.log_prefixlen, randombits_ptrs[i], edge_ptrs[i], entry.num_edge);
        }
        else{
            uint64_t gridSize = (entry.num_edge + blockSize - 1) / blockSize;
            uint16_t cdabcd = (uint16_t) round((c+d) * (1 << 16) - 0.5);
            generate_randombits_src<<<gridSize, blockSize, 0, streams[i % streams.size()]>>>(entry.src_vid_start, cdabcd, entry.log_n - entry.log_prefixlen, randombits_ptrs[i], edge_ptrs[i], entry.num_edge);

            
            uint16_t bab = (uint16_t) round(b/(a+b) * (1 << 16) - 0.5);
            uint16_t dcd = (uint16_t) round(d/(c+d) * (1 << 16) - 0.5);
            int randombits_used = entry.num_edge * (entry.log_n - entry.log_prefixlen);
            randombits_used = randombits_used + 16 - randombits_used % 16;//align to 16 byte
            generate_randombits_dst<<<gridSize, blockSize, 0, streams[i % streams.size()]>>>(entry.dst_vid_start, bab, dcd, entry.log_n, (uint16_t*)(randombits_ptrs[i]) + randombits_used, edge_ptrs[i], entry.num_edge);
        }
    }


    if(hipPeekAtLastError() != hipSuccess){
        std::cerr << "Error processing workloads" << std::endl;
        std::cerr << hipGetErrorString(hipPeekAtLastError()) << std::endl;
        exit(1);
    }

    hipDeviceptr_t dvptr = hipDeviceptr_t(edge_arr_device);

    hipMemcpyDtoH(edge_arr_host_list[hostMemIdx], dvptr, total_edges * EDGE_BYTE);
    writer.write_async(filename, (char*) edge_arr_host_list[hostMemIdx], total_edges * EDGE_BYTE);
    hostMemIdx = (hostMemIdx + 1) % edge_arr_host_list.size();
}